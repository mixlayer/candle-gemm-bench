// Only supports a single scalar scale for the FP8 (B) operand.
// Preserves: explicit A/B swap+transpose to enable TMA epilogue.

#include <cstdint>
#include <hip/hip_runtime.h>

#include "cutlass/cutlass.h"
#include "cute/tensor.hpp"
#include "cutlass/tensor_ref.h"

#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/kernel/gemm_universal.hpp"

#include "cutlass/numeric_types.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/device_memory.h"

using namespace cute;

// ---------------- Types & kernel config (unchanged core) --------------------
using MmaType   = cutlass::bfloat16_t;              // wide type used by tensor cores
using QuantType = cutlass::float_e4m3_t;        // narrow FP8 on B
constexpr int TileShapeK = 128 * 8 / sizeof_bits<MmaType>::value;

// A
using ElementA   = MmaType;
using LayoutA    = cutlass::layout::RowMajor;
constexpr int AlignmentA = 128 / cutlass::sizeof_bits<ElementA>::value;

// B (FP8)
using ElementB   = QuantType;
using LayoutB    = cutlass::layout::ColumnMajor;
constexpr int AlignmentB = 128 / cutlass::sizeof_bits<ElementB>::value;

// Transposed for explicit swap path
using LayoutA_Transpose = typename cutlass::layout::LayoutTranspose<LayoutA>::type;
using LayoutB_Transpose = typename cutlass::layout::LayoutTranspose<LayoutB>::type;

// C/D
using ElementC = cutlass::bfloat16_t;
using ElementD   = ElementC;
using LayoutC    = cutlass::layout::RowMajor;
using LayoutD    = LayoutC;
constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;
constexpr int AlignmentD = 128 / cutlass::sizeof_bits<ElementD>::value;

// Core kernel params
using ElementAccumulator = float;
using ElementCompute     = float;
using ArchTag            = cutlass::arch::Sm90;
// using ArchTag            = cutlass::arch::Sm120;
using OperatorClass      = cutlass::arch::OpClassTensorOp;
using TileShape          = Shape<_128,_128,_128>;
using ClusterShape       = Shape<_2,_1,_1>;

// using KernelSchedule = cutlass::gemm::KernelTmaWarpSpecializedCooperativeMixedInput;
using KernelSchedule     = cutlass::gemm::KernelTmaWarpSpecializedCooperative;
// using KernelSchedule = cutlass::gemm::KernelTmaWarpSpecializedCooperativeFP8FastAccum;

using EpilogueSchedule   = cutlass::epilogue::TmaWarpSpecializedCooperative;
using EpilogueTileType   = cutlass::epilogue::collective::EpilogueTileAuto;

using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
  ArchTag, cutlass::arch::OpClassTensorOp,
  TileShape, ClusterShape,
  EpilogueTileType,
  ElementAccumulator, ElementAccumulator,
  // explicit swap+transpose: epilogue sees transposed C/D
  ElementC, typename cutlass::layout::LayoutTranspose<LayoutC>::type, AlignmentC,
  ElementD, typename cutlass::layout::LayoutTranspose<LayoutD>::type, AlignmentD,
  EpilogueSchedule
>::CollectiveOp;

// Mainloop: ConvertOnly (no per-tile scales/zeros)
using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
  ArchTag, OperatorClass,
  ElementB, LayoutB_Transpose, AlignmentB,   // B first (swapped)
  ElementA, LayoutA_Transpose, AlignmentA,
  ElementAccumulator,
  TileShape, ClusterShape,
  cutlass::gemm::collective::StageCountAutoCarveout<
    static_cast<int>(sizeof(typename CollectiveEpilogue::SharedStorage))>,
  KernelSchedule
>::CollectiveOp;

using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
  Shape<int,int,int,int>,  // (M,N,K,L) placeholder
  CollectiveMainloop,
  CollectiveEpilogue
>;
using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

// Strides (packed, derived from layouts)
using StrideA = cutlass::detail::TagToStrideA_t<LayoutA>;
using StrideB = cutlass::detail::TagToStrideB_t<LayoutB>;
using StrideC = typename GemmKernel::StrideC;
using StrideD = typename GemmKernel::StrideD;

extern "C" {

// 1 = supported on current device (Hopper / SM90*), else 0.
int cutlass_hopper_mixed_dtype_is_supported() {
#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)
  int dev = 0;
  hipDeviceProp_t props{};
  if (hipGetDevice(&dev) != hipSuccess) return 0;
  if (hipGetDeviceProperties(&props, dev) != hipSuccess) return 0;
  return (props.major == 9) ? 1 : 0;
#else
  return 0;
#endif
}

const char* cutlass_status_string(int status_code) {
  return cutlassGetStatusString(static_cast<cutlass::Status>(status_code));
}

// Single entry point: tensor-wide scale for FP8 B.
// All pointers are *device* pointers. If beta==0, C may be null.
// Layouts must match the original example (A: [M,K,L] row-major, B: [K,N,L] col-major, C/D: [M,N,L] row-major).
int cutlass_hopper_fp8_gemm_run_scalar(
    int m, int n, int k, int batch_l,
    float scale_b,      // tensor-wide scale for FP8 B
    float alpha, float beta,
    const void* A,      // half
    const void* B,      // e4m3
    const void* C,      // half (nullable when beta==0)
    void* D,            // half (output)
    void* stream_void   // hipStream_t or hipStream_t (opaque). null => default stream
) {
#if !defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)
  (void)m;(void)n;(void)k;(void)batch_l;(void)scale_b;(void)alpha;(void)beta;
  (void)A;(void)B;(void)C;(void)D;(void)stream_void;
  return static_cast<int>(cutlass::Status::kErrorNotSupported);
#else
  hipStream_t stream = reinterpret_cast<hipStream_t>(stream_void);

  // Shapes and packed strides (note C/D reversed due to explicit swap+transpose)
  StrideA stride_A = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(m, k, batch_l));
  StrideB stride_B = cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(n, k, batch_l));
  StrideC stride_C = cutlass::make_cute_packed_stride(StrideC{}, cute::make_shape(n, m, batch_l));
  StrideD stride_D = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(n, m, batch_l));

  // Fold the FP8 tensor-wide scale into alpha (A*(sB) == s*(A*B)).
  float alpha_eff = alpha * scale_b;

  typename Gemm::Arguments args{
    cutlass::gemm::GemmUniversalMode::kGemm,
    {n, m, k, batch_l},   // swapped problem (N, M, K, L)
    {reinterpret_cast<const ElementB*>(B), stride_B,
     reinterpret_cast<const ElementA*>(A), stride_A},
    {{alpha_eff, beta},
     reinterpret_cast<const ElementC*>(C), stride_C,
     reinterpret_cast<ElementD*>(D), stride_D}
  };

  Gemm gemm;
  size_t ws = Gemm::get_workspace_size(args);
  cutlass::device_memory::allocation<uint8_t> workspace(ws);

  cutlass::Status st;
  st = gemm.can_implement(args);                      if (st != cutlass::Status::kSuccess) return static_cast<int>(st);
  st = gemm.initialize(args, workspace.get(), stream);if (st != cutlass::Status::kSuccess) return static_cast<int>(st);
  st = gemm.run(stream);                              return static_cast<int>(st);
#endif
}

} // extern "C"
